#include "hip/hip_runtime.h"
#include "cumexhelp.h"
#define EXPORT_FCNS
#ifndef SHRHELP
    #include "shrhelp.h"
#endif
#include "mat_cudnn_test.h"

#define cudaErrCheck(stat) { cudaErrCheck_((stat)); }
void cudaErrCheck_(hipError_t stat) {
    if (stat != hipSuccess) {
        mexPrintf("CUDA Error: %s\n", hipGetErrorString(stat));
        mexErrMsgTxt("CUDA Error");
    }
}
#define cudnnErrCheck(stat) { cudnnErrCheck_((stat)); }
void cudnnErrCheck_(hipdnnStatus_t stat) {
    if (stat != HIPDNN_STATUS_SUCCESS) {
        mexPrintf( "cuDNN Error: %s\n", hipdnnGetErrorString(stat));
        mexErrMsgTxt("cuDNN Error");
    }
}
__global__ void initGPUData_ker(float *data, int numElements, float value) {
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < numElements) {
      data[tid] = value;
   }
}
void initGPUData(float *data, int numElements, float value) {
   dim3 gridDim;
   dim3 blockDim;
   
   blockDim.x = 1024;
   gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;
   
   initGPUData_ker <<< gridDim, blockDim >>> (data, numElements, value);
}
// void GET_GPU_CONST_PTR(mxArray *arrayPtr,float const *dataPtr)
// {
//     dataPtr=(float const *)(mxGPUGetDataReadOnly (mxGPUCreateFromMxArray(arrayPtr)));
// }
// void GET_GPU_PTR(mxArray *arrayPtr,float *dataPtr)
// {
//     dataPtr=(float *)(mxGPUGetData(mxGPUCreateFromMxArray(arrayPtr)));
// }

EXPORTED_FUNCTION void MAT_CUDNN_test(void* x)
{
    int(*seqLength)=10;
    int (*inputSize)=128;
    int (*miniBatch)=64;
    hipdnnHandle_t cudnnHandle;
    cudnnErrCheck(hipdnnCreate(&cudnnHandle));
    cudaErrCheck(hipMalloc((void**)&x,(*seqLength) * (*inputSize) * (*miniBatch) * sizeof(float)));
    hipdnnDestroy(cudnnHandle);
    hipFree(x);
}
EXPORTED_FUNCTION void MAT_CUDNN_RNN_LSTM_FF(mxArray const *ax,mxArray const *aw,mxArray *ah,mxArray *ac,int *hiddenSize,int *miniBatch，int *inputSize，int *seqLength,void *reserveSpace)
{
   //int(*seqLength)=20;
   int numLayers=1;
   //int hiddenSize=256;
   //int (*inputSize)=128;
   //int (*miniBatch)=64;
   float dropout=0.0;
   bool bidirectional=0;
   int mode=2;
   hipdnnHandle_t cudnnHandle;   
   cudnnErrCheck(hipdnnCreate(&cudnnHandle));
   float const *x=(float const *)mxGPUGetDataReadOnly(mxGPUCreateFromMxArray(ax));
   void *hx = NULL;
   void *cx = NULL;
   void *y;
   void *hy = NULL;
   void *cy = NULL;
   cudaErrCheck(hipMalloc((void**)&hx, numLayers * (*hiddenSize) * (*miniBatch) * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&cx, numLayers * (*hiddenSize) * (*miniBatch) * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&y,(*seqLength) * (*hiddenSize) * (*miniBatch) * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&hy, numLayers * (*hiddenSize) * (*miniBatch) * (bidirectional ? 2 : 1) * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&cy, numLayers * (*hiddenSize) * (*miniBatch) * (bidirectional ? 2 : 1) * sizeof(float)));
   hipdnnTensorDescriptor_t *xDesc, *yDesc;
   hipdnnTensorDescriptor_t hxDesc, cxDesc;
   hipdnnTensorDescriptor_t hyDesc, cyDesc;
   xDesc = (hipdnnTensorDescriptor_t*)malloc((*seqLength) * sizeof(hipdnnTensorDescriptor_t));
   yDesc = (hipdnnTensorDescriptor_t*)malloc((*seqLength) * sizeof(hipdnnTensorDescriptor_t));
   int dimA[3];
   int strideA[3];
   for (int i = 0; i <(*seqLength); i++) {
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&xDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&yDesc[i]));
   
      dimA[0] = (*miniBatch);
      dimA[1] = (*inputSize);
      dimA[2] = 1;
     
      strideA[0] = dimA[2] * dimA[1];
      strideA[1] = dimA[2];
      strideA[2] = 1;

      cudnnErrCheck(hipdnnSetTensorNdDescriptor(xDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
      
      dimA[0] = (*miniBatch);
      dimA[1] = bidirectional ? (*hiddenSize) * 2 : (*hiddenSize);
      dimA[2] = 1;

      strideA[0] = dimA[2] * dimA[1];
      strideA[1] = dimA[2];
      strideA[2] = 1;
      
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(yDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   }
   
   
   dimA[0] = numLayers * (bidirectional ? 2 : 1);
   dimA[1] = (*miniBatch);
   dimA[2] = (*hiddenSize);
   
   strideA[0] = dimA[2] * dimA[1];
   strideA[1] = dimA[2];
   strideA[2] = 1;
   
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&hxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&cxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&hyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&cyDesc));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(hxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(cxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(hyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(cyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   unsigned long long seed = 1337ull; // Pick a seed.
   
   hipdnnDropoutDescriptor_t dropoutDesc;
   cudnnErrCheck(hipdnnCreateDropoutDescriptor(&dropoutDesc));
   
   // How much memory does dropout need for states?
   // These states are used to generate random numbers internally
   // and should not be freed until the RNN descriptor is no longer used
   size_t stateSize;
   void *states;
   cudnnErrCheck(hipdnnDropoutGetStatesSize(cudnnHandle, &stateSize));
   
   cudaErrCheck(hipMalloc(&states, stateSize));
   
   cudnnErrCheck(hipdnnSetDropoutDescriptor(dropoutDesc, 
                             cudnnHandle,
                             dropout, 
                             states, 
                             stateSize, 
                             seed));
                             
   // -------------------------   
   // Set up the RNN descriptor
   // -------------------------
   hipdnnRNNDescriptor_t rnnDesc;
   hipdnnRNNMode_t RNNMode;
   
   cudnnErrCheck(hipdnnCreateRNNDescriptor(&rnnDesc));
   
   if      (mode == 0) RNNMode = HIPDNN_RNN_RELU;
   else if (mode == 1) RNNMode = HIPDNN_RNN_TANH;
   else if (mode == 2) RNNMode = HIPDNN_LSTM;
   else if (mode == 3) RNNMode = HIPDNN_GRU;
      
   cudnnErrCheck(hipdnnSetRNNDescriptor(rnnDesc,
                                       (*hiddenSize), 
                                       numLayers, 
                                       dropoutDesc,
                                       HIPDNN_LINEAR_INPUT, // We can also skip the input matrix transformation
                                       bidirectional ? HIPDNN_BIDIRECTIONAL : HIPDNN_UNIDIRECTIONAL, 
                                       RNNMode, 
                                       HIPDNN_DATA_FLOAT));
   void *w;   
   hipdnnFilterDescriptor_t wDesc;
   cudnnErrCheck(hipdnnCreateFilterDescriptor(&wDesc));   
   size_t weightsSize;
   cudnnErrCheck(hipdnnGetRNNParamsSize(cudnnHandle, rnnDesc, xDesc[0], &weightsSize, HIPDNN_DATA_FLOAT));
   
   int dimW[3];   
   dimW[0] =  weightsSize / sizeof(float);
   dimW[1] = 1;
   dimW[2] = 1;
      
   cudnnErrCheck(hipdnnSetFilterNdDescriptor(wDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, dimW));   
   
   cudaErrCheck(hipMalloc((void**)&w,  weightsSize));
   void *workspace;
   size_t workSize;
   size_t reserveSize;
   cudnnErrCheck(hipdnnGetRNNWorkspaceSize(cudnnHandle, rnnDesc,(*seqLength), xDesc, &workSize));
   // Only needed in training, shouldn't be touched between passes.
   cudnnErrCheck(hipdnnGetRNNTrainingReserveSize(cudnnHandle, rnnDesc,(*seqLength), xDesc, &reserveSize));
    
   cudaErrCheck(hipMalloc((void**)&workspace, workSize));
   cudaErrCheck(hipMalloc((void**)&reserveSpace, reserveSize));
   //if (hx != NULL) initGPUData((float*)hx, numLayers * hiddenSize * (*miniBatch) * (bidirectional ? 2 : 1), 1.f);
   //if (cx != NULL) initGPUData((float*)cx, numLayers * hiddenSize * (*miniBatch) * (bidirectional ? 2 : 1), 1.f);
   int numLinearLayers = 0;
   if (RNNMode == HIPDNN_RNN_RELU || RNNMode == HIPDNN_RNN_TANH) {
      numLinearLayers = 2;
   }
   else if (RNNMode == HIPDNN_LSTM) {
      numLinearLayers = 8;
   }
   else if (RNNMode == HIPDNN_GRU) {
      numLinearLayers = 6;
   }
   for (int layer = 0; layer < numLayers * (bidirectional ? 2 : 1); layer++) {
      for (int linLayerID = 0; linLayerID < numLinearLayers; linLayerID++) {
         hipdnnFilterDescriptor_t linLayerMatDesc;
         cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerMatDesc));
         float *linLayerMat;
         
         cudnnErrCheck(hipdnnGetRNNLinLayerMatrixParams( cudnnHandle,
                                                        rnnDesc,  
                                                        layer,
                                                        xDesc[0], 
                                                        wDesc, 
                                                        w,
                                                        linLayerID,  
                                                        linLayerMatDesc, 
                                                        (void**)&linLayerMat));
         
         hipdnnDataType_t dataType;
         hipdnnTensorFormat_t format;
         int nbDims;
         int filterDimA[3];
         cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerMatDesc,
                                                  3,
                                                  &dataType,
                                                  &format,
                                                  &nbDims,
                                                  filterDimA));
                                                  
         initGPUData(linLayerMat, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f / (float)(filterDimA[0] * filterDimA[1] * filterDimA[2]));                                                 

         cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerMatDesc));         
         
         hipdnnFilterDescriptor_t linLayerBiasDesc;
         cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerBiasDesc));
         float *linLayerBias;
         
         cudnnErrCheck(hipdnnGetRNNLinLayerBiasParams( cudnnHandle,
                                                        rnnDesc,  
                                                        layer,
                                                        xDesc[0], 
                                                        wDesc, 
                                                        w,
                                                        linLayerID,  
                                                        linLayerBiasDesc, 
                                                        (void**)&linLayerBias));
         
         cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerBiasDesc,
                                                  3,
                                                  &dataType,
                                                  &format,
                                                  &nbDims,
                                                  filterDimA));
                                                  
         initGPUData(linLayerBias, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f);
                                                  
         cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerBiasDesc));
      }
   }
   cudaErrCheck(hipDeviceSynchronize());
   
   cudnnErrCheck(hipdnnRNNForwardTraining(cudnnHandle, 
                                         rnnDesc, 
                                        (*seqLength),                                          
                                         xDesc, 
                                         x, 
                                         hxDesc,
                                         hx, 
                                         cxDesc, 
                                         cx, 
                                         wDesc, 
                                         w, 
                                         yDesc,  
                                         y, 
                                         hyDesc, 
                                         hy, 
                                         cyDesc, 
                                         cy, 
                                         workspace, 
                                         workSize,
                                         reserveSpace, 
                                         reserveSize));



   hipFree(hx);
   hipFree(cx);
   hipFree(y);
   hipFree(hy);
   hipFree(cy);
   hipFree(workspace);
   hipFree(reserveSpace);
   hipFree(w);
   hipdnnDestroy(cudnnHandle);

}
void mexFunction( int nlhs, mxArray *plhs[],
        int nrhs, const mxArray*prhs[] )
{
}
