#include"mat_cudnn.h"
void GET_GPU_CONST_PTR(mxArray const *arrayPtr,float const *dataPtr)
{
    dataPtr=(float const *)(mxGPUGetData(mxGPUCreateFromMxArray(arrayPtr)));
}
void GET_GPU_PTR(mxArray const *arrayPtr,float *dataPtr)
{
    dataPtr=(float *)(mxGPUGetData(mxGPUCreateFromMxArray(arrayPtr)));
}
void MAT_CUDNN_LSTM_FF(mxArray const *x_array,mxArray const *w_array,void **reserveSpace,int* minibatch,int* hiddenSize,int* inputSize,int* seqLength )
{   // -------------------------   
   // Create cudnn context
   // -------------------------  
   mxInitGPU();
   hipdnnHandle_t cudnnHandle;   
   cudnnErrCheck(hipdnnCreate(&cudnnHandle));

   hipdnnTensorDescriptor_t *xDesc, *yDesc, *dxDesc, *dyDesc;
   hipdnnTensorDescriptor_t hxDesc, cxDesc;
   hipdnnTensorDescriptor_t hyDesc, cyDesc;
   hipdnnTensorDescriptor_t dhxDesc, dcxDesc;
   hipdnnTensorDescriptor_t dhyDesc, dcyDesc;
   
   xDesc = (hipdnnTensorDescriptor_t*)malloc(*seqLength * sizeof(hipdnnTensorDescriptor_t));
   yDesc = (hipdnnTensorDescriptor_t*)malloc(*seqLength * sizeof(hipdnnTensorDescriptor_t));
   dxDesc = (hipdnnTensorDescriptor_t*)malloc(*seqLength * sizeof(hipdnnTensorDescriptor_t));
   dyDesc = (hipdnnTensorDescriptor_t*)malloc(*seqLength * sizeof(hipdnnTensorDescriptor_t));
   
   int dimA[3];
   int strideA[3];
   // In this example dimA[1] is constant across the whole sequence
   // This isn't required, all that is required is that it does not increase.
   for (int i = 0; i < *seqLength; i++) {
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&xDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&yDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&dxDesc[i]));
      cudnnErrCheck(hipdnnCreateTensorDescriptor(&dyDesc[i]));
   
      dimA[0] = *miniBatch;
      dimA[1] = *inputSize;
      dimA[2] = 1;
     
      strideA[0] = dimA[2] * dimA[1];
      strideA[1] = dimA[2];
      strideA[2] = 1;

      cudnnErrCheck(hipdnnSetTensorNdDescriptor(xDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(dxDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
      
      dimA[0] = *miniBatch;
      dimA[1] = *hiddenSize;
      dimA[2] = 1;

      strideA[0] = dimA[2] * dimA[1];
      strideA[1] = dimA[2];
      strideA[2] = 1;
      
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(yDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
      cudnnErrCheck(hipdnnSetTensorNdDescriptor(dyDesc[i], HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   }
   dimA[0] = 1;
   dimA[1] = *miniBatch;
   dimA[2] = *hiddenSize;
   
   strideA[0] = dimA[2] * dimA[1];
   strideA[1] = dimA[2];
   strideA[2] = 1;
   
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&hxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&cxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&hyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&cyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcxDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dhyDesc));
   cudnnErrCheck(hipdnnCreateTensorDescriptor(&dcyDesc));
   
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(hxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(cxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(hyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(cyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcxDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dhyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   cudnnErrCheck(hipdnnSetTensorNdDescriptor(dcyDesc, HIPDNN_DATA_FLOAT, 3, dimA, strideA));
   // -------------------------
   // Set up the dropout descriptor (needed for the RNN descriptor)
   // -------------------------
   unsigned long long seed = 1337ull; // Pick a seed.
   
   hipdnnDropoutDescriptor_t dropoutDesc;
   cudnnErrCheck(hipdnnCreateDropoutDescriptor(&dropoutDesc));
   
   // How much memory does dropout need for states?
   // These states are used to generate random numbers internally
   // and should not be freed until the RNN descriptor is no longer used
   size_t stateSize;
   void *states;
   cudnnErrCheck(hipdnnDropoutGetStatesSize(cudnnHandle, &stateSize));
   
   cudaErrCheck(hipMalloc(&states, stateSize));
   float dropout=0;
   cudnnErrCheck(hipdnnSetDropoutDescriptor(dropoutDesc, 
                             cudnnHandle,
                             dropout, 
                             states, 
                             stateSize, 
                             seed));
                             
   // -------------------------   
   // Set up the RNN descriptor
   // -------------------------
   hipdnnRNNDescriptor_t rnnDesc;
   hipdnnRNNMode_t RNNMode;
   
   cudnnErrCheck(hipdnnCreateRNNDescriptor(&rnnDesc));
   
   RNNMode = HIPDNN_LSTM;
      
   cudnnErrCheck(hipdnnSetRNNDescriptor(rnnDesc,
                                       hiddenSize, 
                                       numLayers, 
                                       dropoutDesc,
                                       HIPDNN_LINEAR_INPUT, // We can also skip the input matrix transformation
                                       HIPDNN_UNIDIRECTIONAL, 
                                       RNNMode, 
                                       HIPDNN_DATA_FLOAT));
   // -------------------------
   // Set up parameters
   // -------------------------
   // This needs to be done after the rnn descriptor is set as otherwise
   // we don't know how many parameters we have to allocate
   void *w;   
   void *dw;   

   hipdnnFilterDescriptor_t wDesc, dwDesc;
   
   cudnnErrCheck(hipdnnCreateFilterDescriptor(&wDesc));
   cudnnErrCheck(hipdnnCreateFilterDescriptor(&dwDesc));
   
   size_t weightsSize;
   cudnnErrCheck(hipdnnGetRNNParamsSize(cudnnHandle, rnnDesc, xDesc[0], &weightsSize, HIPDNN_DATA_FLOAT));
   
   int dimW[3];   
   dimW[0] =  weightsSize / sizeof(float);
   dimW[1] = 1;
   dimW[2] = 1;
      
   cudnnErrCheck(hipdnnSetFilterNdDescriptor(wDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, dimW));   
   cudnnErrCheck(hipdnnSetFilterNdDescriptor(dwDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, dimW));   
   
   cudaErrCheck(hipMalloc((void**)&w,  weightsSize));
   cudaErrCheck(hipMalloc((void**)&dw, weightsSize));
   
   
   // -------------------------
   // Set up work space and reserved memory
   // -------------------------   
   void *workspace;
   
   size_t workSize;
   size_t reserveSize;

   // Need for every pass
   cudnnErrCheck(hipdnnGetRNNWorkspaceSize(cudnnHandle, rnnDesc, seqLength, xDesc, &workSize));
   // Only needed in training, shouldn't be touched between passes.
   cudnnErrCheck(hipdnnGetRNNTrainingReserveSize(cudnnHandle, rnnDesc, seqLength, xDesc, &reserveSize));
    
   cudaErrCheck(hipMalloc((void**)&workspace, workSize));
   cudaErrCheck(hipMalloc((void**)&reserveSpace, reserveSize));
   // Weights
   int numLinearLayers = 0;
   for (int layer = 0; layer < numLayers * (bidirectional ? 2 : 1); layer++) {
      for (int linLayerID = 0; linLayerID < numLinearLayers; linLayerID++) {
         hipdnnFilterDescriptor_t linLayerMatDesc;
         cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerMatDesc));
         float *linLayerMat;
         
         cudnnErrCheck(hipdnnGetRNNLinLayerMatrixParams( cudnnHandle,
                                                        rnnDesc,  
                                                        layer,
                                                        xDesc[0], 
                                                        wDesc, 
                                                        w,
                                                        linLayerID,  
                                                        linLayerMatDesc, 
                                                        (void**)&linLayerMat));
         
         hipdnnDataType_t dataType;
         hipdnnTensorFormat_t format;
         int nbDims;
         int filterDimA[3];
         cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerMatDesc,
                                                  3,
                                                  &dataType,
                                                  &format,
                                                  &nbDims,
                                                  filterDimA));
                                                  
         initGPUData(linLayerMat, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f / (float)(filterDimA[0] * filterDimA[1] * filterDimA[2]));                                                 

         cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerMatDesc));         
         
         hipdnnFilterDescriptor_t linLayerBiasDesc;
         cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerBiasDesc));
         float *linLayerBias;
         
         cudnnErrCheck(hipdnnGetRNNLinLayerBiasParams( cudnnHandle,
                                                        rnnDesc,  
                                                        layer,
                                                        xDesc[0], 
                                                        wDesc, 
                                                        w,
                                                        linLayerID,  
                                                        linLayerBiasDesc, 
                                                        (void**)&linLayerBias));
         
         cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerBiasDesc,
                                                  3,
                                                  &dataType,
                                                  &format,
                                                  &nbDims,
                                                  filterDimA));
                                                  
         initGPUData(linLayerBias, filterDimA[0] * filterDimA[1] * filterDimA[2], 1.f);
                                                  
         cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerBiasDesc));
      }
   }
   // *********************************************************************************************************
   // At this point all of the setup is done. We now need to pass through the RNN.
   // *********************************************************************************************************
   
  
   
   cudaErrCheck(hipDeviceSynchronize());
   
   hipEvent_t start, stop;
   float timeForward, timeBackward1, timeBackward2;
   cudaErrCheck(hipEventCreate(&start));
   cudaErrCheck(hipEventCreate(&stop));
   
   cudaErrCheck(hipEventRecord(start));   

   // If we're not training we use this instead
   // cudnnErrCheck(hipdnnRNNForwardInference(cudnnHandle, 
                                         // rnnDesc, 
                                         // xDesc, 
                                         // x, 
                                         // hxDesc,
                                         // hx, 
                                         // cxDesc, 
                                         // cx, 
                                         // wDesc, 
                                         // w, 
                                         // yDesc,  
                                         // y, 
                                         // hyDesc, 
                                         // hy, 
                                         // cyDesc, 
                                         // cy, 
                                         // workspace, 
                                         // workSize));

   cudnnErrCheck(hipdnnRNNForwardTraining(cudnnHandle, 
                                         rnnDesc, 
                                         seqLength,                                          
                                         xDesc, 
                                         x, 
                                         hxDesc,
                                         hx, 
                                         cxDesc, 
                                         cx, 
                                         wDesc, 
                                         w, 
                                         yDesc,  
                                         y, 
                                         hyDesc, 
                                         hy, 
                                         cyDesc, 
                                         cy, 
                                         workspace, 
                                         workSize,
                                         reserveSpace, 
                                         reserveSize));
   

}